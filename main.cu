#include "hip/hip_runtime.h"

/**
 * @license MIT
 * @author Marco Lourenço
 * @date 14.11.17.
 */

// cuda_runtime include not needed by nvcc, only to help IDE
#include <hip/hip_runtime.h>
#include <iostream>

#include "enum.h"
#include "cuda_helper.h"

__device__ void giveTheColorPlz(const Color &c) {
  Color kernel_color = Color::red;
  bool tmp = (c == kernel_color);
  printf("device fct param is red: %d\n", tmp);
}

__global__ void testEnumKernel(Color *color_device,
                               const Animal *animal_device,
                               const Mammal *mammal_device,
                               const ManagedAnimal *managed_animal) {
  printf("animal casted: %u\n", static_cast<ushort>(*animal_device));
  printf("animal not explicitly casted: %u\n", *animal_device);
  printf("mammal casted: %u\n", static_cast<ushort>(*mammal_device));
  printf("mammal not explicitly casted: %u\n", *mammal_device);
  printf("color not explicitly casted: %u\n", *color_device);

  bool tmp = (*color_device == Color::red);
  printf("color equals red: %d\n", tmp);

  *color_device = Color::blue;
  printf("color set to blue\n");
  tmp = (*color_device == Color::red);
  printf("color equals red: %d\n", tmp);

  printf("color not explicitly casted: %u\n", *color_device);

  giveTheColorPlz(Color::red);

  tmp = (*(managed_animal->animal) == Animal::cat);
  printf("managed animal is cat: %d\n", tmp);
}

int main() {

  //--- Without UNIFIED MEMORY ---

  Color red_host = Color::red;
  Animal cat_host = Animal::cat;
  Mammal deer_host = Mammal::deer;

  Color *red_device;
  hipMalloc(&red_device, sizeof(Color));
  hipMemcpy(red_device, &red_host, sizeof(Color), hipMemcpyHostToDevice);

  Animal *cat_device;
  hipMalloc(&cat_device, sizeof(Animal));
  hipMemcpy(cat_device, &cat_host, sizeof(Animal), hipMemcpyHostToDevice);

  Mammal *deer_device;
  hipMalloc(&deer_device, sizeof(Mammal));
  hipMemcpy(deer_device, &deer_host, sizeof(Mammal), hipMemcpyHostToDevice);

  //--- With UNIFIED MEMORY ---

  // ManagedAnimal managed_animal; --> THIS WILL NOT WORK
  ManagedAnimal *managed_animal = new ManagedAnimal;
  *(managed_animal->animal) = Animal::cat;

  testEnumKernel <<< 1, 1 >>>
      (red_device, cat_device, deer_device, managed_animal);
  CUDA_SAFE_CALL(hipDeviceSynchronize()) // this should NOT give a CUDA error

  return EXIT_SUCCESS;
}